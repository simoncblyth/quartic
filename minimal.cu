// nvcc -arch=sm_30 minimal.cu -run ; rm a.out 
#define CUDA_BOTH __device__ __host__

#include <hip/hip_runtime.h>
#include <stdio.h>


CUDA_BOTH
static double cubic_root(double x) ;

__global__ void fquartic(float a4, float a3, float a2, float a1, float a0)
{
      printf("fquartic %d, a4=%g a3=%g a2=%g a1=%g a0=%g \n", threadIdx.x, a4,a3,a2,a1,a0 );
}

__global__ void dquartic(double a4, double a3, double a2, double a1, double a0)
{
      double a[5] ; 
      a[4] = a4 ; 
      a[3] = a3 ;
      a[2] = a2 ;
      a[1] = a1 ;
      a[0] = a0 ;

      printf("dquartic %d, a[4]=%g a[3]=%g a[2]=%g a[1]=%g a[0]=%g \n", threadIdx.x, a[4],a[3],a[2],a[1],a[0] );
      printf("cubic_root(27) %g \n", cubic_root(27) );
}

CUDA_BOTH
static double cubic_root(double x) {
    const double t = pow(fabs(x), 1.0 / 3.0);
    return x >= 0.0 ? t : -t;
}


int main()
{
    double a[5] ; 
    //float a[5] ; 

    a[4] = 1. ; 
    a[3] = -10. ;
    a[2] =  35. ;
    a[1] = -50. ;
    a[0] = 24. ; 


    dquartic<<<1, 1>>>( a[4], a[3], a[2], a[1], a[0] );

    hipDeviceReset();
    return 0;
}



