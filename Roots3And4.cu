#include "hip/hip_runtime.h"

// nvcc -arch=sm_30 Roots3And4.cu -run ; rm a.out 

#include "Roots3And4.h"
#include <stdio.h>


__global__ void solve_real_quartic(double a4, double a3, double a2, double a1, double a0)
{
    printf("..solve_real_quartic %d, a4=%g a3=%g a2=%g a1=%g a0=%g \n", threadIdx.x, a4,a3,a2,a1,a0 );
    printf("cbrt(27) %g \n", cbrt(27.0) );

    double poly[5];
    poly[4] = a4 ; 
    poly[3] = a3 ;
    poly[2] = a2 ;
    poly[1] = a1 ;
    poly[0] = a0 ;
 
    double sols[4];

    printf("..chk %d \n", 0 );
    const int num_sols = SolveQuartic( poly, sols);

    printf("..num_sols %d \n", num_sols);
    for (int i = 0; i < num_sols; ++i) printf("%f\n", sols[i]);
}



int main()
{
    double a[5] ; 
    a[4] = 1. ; 
    a[3] = -10. ;
    a[2] =  35. ;
    a[1] = -50. ;
    a[0] = 24. ; 

    size_t limit = 0;

    hipDeviceGetLimit(&limit, hipLimitStackSize);
    printf("hipLimitStackSize: %u\n", (unsigned)limit);

    // hipDeviceSetLimit(hipLimitStackSize, limit*2 );


    solve_real_quartic<<<1, 1>>>( a[4], a[3], a[2], a[1], a[0] );

    hipDeviceReset();
    return 0;
}




